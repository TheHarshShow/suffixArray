#include "hip/hip_runtime.h"
#include "suffixArray.cuh"
#include <iostream>

__global__ void assignIndexes(size_t l, char* seq, uint32_t* indexes){
    int tx = threadIdx.x;
    int bx = blockIdx.x;
    int bs = blockDim.x;
    int gs = gridDim.x;

    for(int i = bx*bs+tx; i < l; i+=bs*gs){
        indexes[i] = i;
    }
    // __syncthreads();

    // if(bx == 0 && tx == 0){
    //     for(int i = 0; i < 20; i++){
    //         printf("%d %c\n", indexes[i], seq[i]);
    //     }
    // }

}

void SuffixArray::Sequence::allocateSequenceArray(size_t n){
    l = n;
    hipMalloc(&sequence, l*sizeof(char));
    hipMalloc(&indexes, l*sizeof(uint32_t));
}

void SuffixArray::Sequence::copyToGPU(char* cpuSequence){
    hipMemcpy(sequence, cpuSequence, l*sizeof(char), hipMemcpyHostToDevice);

    int numBlocks = 1024; // i.e. number of thread blocks on the GPU
    int blockSize = 512; 

    assignIndexes<<<numBlocks, blockSize>>>(l, sequence, indexes);
}

#define HANDLE_GPU_ERROR(ans)		\
{									\
	hipError_t errorNum = ans;		\
	if (errorNum != hipSuccess)	\
	{								\
		std::cout 	<< std::dec <<	hipGetErrorString( errorNum ) << " in " << __FILE__ << " at line " << __LINE__ << std::endl; \
		exit(-1);					\
	}								\
}

void SuffixArray::Sequence::copyToCPU(uint32_t* cpuIndexes){
    // printf("L:::: %d\n", l);
    // for(size_t i = 0; i < 20; i++){
    //     // std::cout << cpuIndexes[i] << std::endl;
    //     printf("%d\n", cpuIndexes[i]);
    // }

    // int err = hipMemcpy(cpuIndexes, indexes, l*sizeof(uint32_t), hipMemcpyDeviceToHost);
    // if (err != hipSuccess) {
    //     fprintf(stderr, "GPU_ERROR: cudaMemCpy failed! %d\n", err);
    //     exit(1);
    // }

    HANDLE_GPU_ERROR( hipMemcpy(cpuIndexes, indexes, l*sizeof(uint32_t), hipMemcpyDeviceToHost) );
    for(int i = 0; i < 20; i++){
        std::cout << cpuIndexes[i] << std::endl;
    }

    // hipMemcpy(cpuIndexes, indexes, l*sizeof(uint32_t), hipMemcpyDeviceToHost);
    // hipMemcpy(cpuIndexes, indexes, (l) * sizeof(uint32_t), hipMemcpyDeviceToHost);
}

void SuffixArray::Sequence::freeSequenceArray(){
    hipFree(sequence);
    hipFree(indexes);
}